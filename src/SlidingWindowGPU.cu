#include "hip/hip_runtime.h"

#include "SlidingWindowGPU.cuh"

#include "sliding_window_kernels.cuh"

#include "my_cufft_utils.hpp"
#include "my_cuda_utils.hpp"
#include "my_utils.hpp"

SlidingWindowGPU::SlidingWindowGPU( 
   const int new_num_samples, 
   const int new_window_size,
   const int new_threads_per_block,
   const int new_seed,
   const mode_select_t new_mode_select,
   const std::string new_filename,
   const bool new_debug ):
      num_samples( new_num_samples ),
      window_size( new_window_size ),
      threads_per_block( new_threads_per_block ),
      seed( new_seed ),
      mode_select( new_mode_select ),
      filename( new_filename ),
      debug( new_debug ) {

   try {
      hipError_t cerror = hipSuccess;         
      try_cuda_func_throw( cerror, hipGetDevice( &device_id ) );

      hipDeviceProp_t deviceProp;
      try_cuda_func_throw( cerror, hipGetDeviceProperties( &deviceProp, device_id ) );

      size_t max_shared_mem_per_block = deviceProp.sharedMemPerBlock;
      dout << __func__ << "(): Max shared memory per block is " <<  max_shared_mem_per_block << " bytes\n";

      stream_ptr = my_make_unique<hipStream_t>();
      try_cudaStreamCreate( stream_ptr.get() );
      dout << __func__ << "(): after hipStreamCreate()\n"; 

      dout << __func__ << "(): num_samples is " << num_samples << "\n";
      dout << __func__ << "(): window_size is " << window_size << "\n";

      size_t max_num_shared_samples_per_block = max_shared_mem_per_block/sizeof(hipfftComplex);
      dout << __func__ << "(): Max number of samples fitting in shared memory per block is " <<  max_num_shared_samples_per_block << "\n";
      
      size_t max_num_shared_windows = max_num_shared_samples_per_block/window_size;
      dout << __func__ << "(): Max number of windows fitting in shared memory per block is " <<  max_num_shared_windows << "\n";
      
      if ( threads_per_block > max_num_shared_windows ) {
         std::cout << "Threads per block, " << threads_per_block << " is more than the max that can fit in shared memory per block: "
            << max_num_shared_windows << "\n";
         threads_per_block = max_num_shared_windows;
         std::cout << "Changing threads per block to max_num_shared_windows: " << threads_per_block << " so that the windows will fit into shared memory.\n\n"; 
      }

      dout << __func__ << "(): threads_per_block is " << threads_per_block << "\n";

      num_blocks = (num_samples + (threads_per_block-1))/threads_per_block;
      dout << __func__ << "(): num_blocks is " << num_blocks << "\n";

      adjusted_num_samples = threads_per_block * num_blocks;
      adjusted_num_sample_bytes = adjusted_num_samples * sizeof( hipfftComplex );

      num_shared_bytes = threads_per_block * window_size * sizeof( hipfftComplex );
      dout << __func__ << "(): number of shared bytes is " << num_shared_bytes << "\n";
      

      num_windowed_samples = num_samples - window_size;
      dout << __func__ << "(): number of windowed samples is "
         << num_windowed_samples << "\n";

      dout << __func__ << "(): adjusted number of samples for allocation is " 
         << adjusted_num_samples << "\n";
      dout << __func__ << "(): adjusted number of sample bytes for hipMemcpyAsync is "
         << adjusted_num_sample_bytes << "\n";

      char* user_env = getenv( "USER" );
      if ( user_env == nullptr ) {
         throw std::runtime_error( std::string{__func__} + 
            "(): Empty USER env. USER environment variable needed for paths to files" ); 
      }
      
      std::string filepath_prefix = "/home/" + std::string{user_env} + "/Sandbox/CUDA/norm_autocorr/";

      filepath = filepath_prefix + filename;

      dout << __func__ << "(): Filepath is " << filepath << "\n";

      window_sums.reserve( adjusted_num_samples );
      window_sums.resize( adjusted_num_samples );
      std::fill( window_sums.begin(), window_sums.end(), make_hipFloatComplex( 0.f, 0.f ) );
      
      samples.reserve( adjusted_num_samples );
      samples.resize( adjusted_num_samples );

      try_cuda_func_throw( cerror, hipHostGetDevicePointer( &d_samples, samples.data(), 0 ) );
      try_cuda_func_throw( cerror, hipHostGetDevicePointer( &d_window_sums, window_sums.data(), 0 ) );
      
      exp_window_sums = new hipfftComplex[num_samples];
      for( int index = 0; index < num_samples; ++index ) {
         exp_window_sums[index] = make_hipFloatComplex(0.f,0.f);
      }

      dout << __func__ << "(): Mode Select is " << get_mode_select_string( mode_select ) << "\n";

   } catch( std::exception& ex ) {
      throw std::runtime_error{
         std::string{__func__} + std::string{"(): "} + ex.what()
      }; 
   }      
} // end of SlidingWindowGPU()


void SlidingWindowGPU::initialize_samples() {
   try {
      std::fill( samples.begin(), samples.end(), make_hipFloatComplex(0.f,0.f) );

      if( mode_select == mode_select_t::Sinusoidal ) {
         dout << __func__ << "(): Sinusoidal Sample Test Selected\n";
         for( size_t index = 0; index < num_samples; ++index ) {
            float t_val_real = AMPLITUDE*sin(2*PI*FREQ*index);
            float t_val_imag = AMPLITUDE*cos(2*PI*FREQ*index);
            samples[index] = make_hipFloatComplex( t_val_real, t_val_imag );
         }

      } else if ( mode_select == mode_select_t::Random ) {
         dout << __func__ << "(): Random Sample Test Selected. Seed is " << seed << "\n";
         gen_cufftComplexes( samples.data(), num_samples, -AMPLITUDE, AMPLITUDE, seed );

      } else if ( mode_select == mode_select_t::Filebased ) {
         dout << __func__ << "(): File-Based Sample Test Selected. File is " << filename << "\n";
         read_binary_file<hipfftComplex>( 
            samples,
            filepath.c_str(),
            num_samples, 
            debug 
         );
      } // end of else-ifs for mode_select

   } catch( std::exception& ex ) {
      throw std::runtime_error{
         std::string{__func__} + std::string{"(): "} + ex.what()
      }; 
   } // end of try      
} // end of void initialize_samples()


void SlidingWindowGPU::gen_expected_window_sums() { 
   cpu_run(); 
}


void SlidingWindowGPU::print_results( const std::string& prefix = "Window Sums: " ) {
   print_cufftComplexes( window_sums.data(), num_samples, prefix.c_str(),  " ",  "\n" );
}


void SlidingWindowGPU::calc_exp_window_sums() {

   // exp_window_sums must already be all zeros
   dout << __func__ << "(): exp_window_sums[0] = { " 
      << exp_window_sums[0].x << ", " << exp_window_sums[0].y << " }\n"; 

   for( int index = 0; index < window_size; ++index ) {
      exp_window_sums[0] = hipCaddf( exp_window_sums[0], samples[index] );
   }

   dout << __func__ << "(): after initial summation, exp_window_sums[0] = { " 
      << exp_window_sums[0].x << ", " << exp_window_sums[0].y << " }\n"; 
      
   dout << __func__ << "(): num_windowed_samples is " << num_windowed_samples << "\n"; 
   for( int index = 1; index < num_windowed_samples; ++index ) {
      hipfftComplex temp = hipCsubf( exp_window_sums[index-1], samples[index-1] );
      exp_window_sums[index] = hipCaddf( temp, samples[index + window_size-1] );
   } 

} // end of calc_exp_window_sums()


void SlidingWindowGPU::cpu_run() {
   try { 
      float cpu_milliseconds = 0.f;
      
      dout << __func__ << "(): num_samples is " << num_samples << "\n";
      
      Time_Point start = Steady_Clock::now();

      calc_exp_window_sums();
      
      Duration_ms duration_ms = Steady_Clock::now() - start;
      cpu_milliseconds = duration_ms.count();

      std::cout << "It took the CPU " << cpu_milliseconds << " milliseconds to process " << num_samples << " samples\n";
      std::cout << "That's a rate of " << ((num_samples*1000.f)/cpu_milliseconds) << " samples processed per second\n\n"; 

   } catch( std::exception& ex ) {
      throw std::runtime_error( std::string{__func__} +  std::string{"(): "} + ex.what() ); 
   }
}


void SlidingWindowGPU::run() {
   try {
      hipError_t cerror = hipSuccess;

      dout << __func__ << "(): num_samples is " << num_samples << "\n"; 
      dout << __func__ << "(): threads_per_block is " << threads_per_block << "\n"; 
      dout << __func__ << "(): num_blocks is " << num_blocks << "\n\n"; 
      
      dout << __func__ << "(): adjusted_num_samples is " << adjusted_num_samples << "\n"; 
      dout << __func__ << "(): adjusted_num_sample_bytes is " << adjusted_num_sample_bytes << "\n"; 

      initialize_samples();
      gen_expected_window_sums();

      if ( debug ) {
         print_cufftComplexes( samples.data(), num_samples, "Samples: ", " ", "\n" ); 
         print_cufftComplexes( exp_window_sums, num_samples, "Expected Window Sums: ", " ", "\n" ); 
      }
      
      float gpu_milliseconds = 0.f;
      Time_Point start = Steady_Clock::now();
      
      //sliding_window_original<<<num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( 
      sliding_window_sh_mem<<<num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( 
         d_window_sums, 
         d_samples,
         window_size,
         num_windowed_samples 
      );

      try_cuda_func_throw( cerror, hipDeviceSynchronize() );
      
      Duration_ms duration_ms = Steady_Clock::now() - start;
      gpu_milliseconds = duration_ms.count();

      float max_diff = 1;
      bool all_close = false;
      if ( debug ) {
         print_results( "Window Sums: " );
         std::cout << "\n"; 
      }
      dout << __func__ << "(): Window Sums Check:\n"; 
      all_close = cufftComplexes_are_close( window_sums.data(), exp_window_sums, num_samples, max_diff, "window sums: ", debug );
      if (!all_close) {
         throw std::runtime_error{ std::string{__func__} + 
            std::string{"(): Mismatch between actual window_sums from GPU and expected window_sums."} };
      }
      dout << "\n"; 
      
      std::cout << "All " << num_samples << " Window Sums matched expected Window Sums. Test Passed.\n\n"; 
      std::cout << "It took the GPU " << gpu_milliseconds 
         << " milliseconds to process " << num_samples 
         << " samples\n";

      std::cout << "That's a rate of " << ( (num_samples*1000.f)/gpu_milliseconds ) << " samples processed per second\n"; 


   } catch( std::exception& ex ) {
      std::cout << __func__ << "(): " << ex.what() << "\n"; 
   }
}


SlidingWindowGPU::~SlidingWindowGPU() {
   dout << __func__ << "() called\n";
   samples.clear();    
   window_sums.clear();

   if ( exp_window_sums ) delete [] exp_window_sums;
   
   if ( stream_ptr ) hipStreamDestroy( *(stream_ptr.get()) );
   dout << __func__ << "() done\n";
}

