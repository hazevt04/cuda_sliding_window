#include "hip/hip_runtime.h"
#include "sliding_window_kernels.cuh"

//////////////////////////////////////
// Calculate sliding window sums
//////////////////////////////////////
__global__ void sliding_window_original(
   hipfftComplex* __restrict__ window_sums, 
   hipfftComplex* const __restrict__ samples, 
   const int window_size, 
   const int num_windowed_samples ) {
   
   // Assuming one stream
   int global_index = 1*(blockIdx.x * blockDim.x) + 1*(threadIdx.x);
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;
   for ( int index = global_index; index < num_windowed_samples; index+=stride ) {
      hipfftComplex t_window_sum = make_hipFloatComplex(0.f, 0.f);

      for ( int w_index = 0; w_index < window_size; ++w_index ) {
         t_window_sum = make_hipFloatComplex( ( t_window_sum.x + samples[index + w_index].x ),
            ( t_window_sum.y + samples[index + w_index].y ) );
      }

      window_sums[index] = make_hipFloatComplex( t_window_sum.x, t_window_sum.y );
   }
} // end of __global__ void sliding_window_original

// Shared Memory Implementation
__global__ void sliding_window_sh_mem( hipfftComplex* __restrict__ window_sums, 
   hipfftComplex* const __restrict__ samples, 
   const int window_size, 
   const int num_windowed_samples ) {

   // Assuming one stream
   int global_index = 1*(blockIdx.x * blockDim.x) + 1*(threadIdx.x);
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;

   __shared__ hipfftComplex sh_samples[window_size * blockDim.x];
   __shared__ hipfftComplex sh_window_sums[blockDim.x];

   for ( int index = global_index; index < num_windowed_samples; index+=stride ) {
      
      for( int w_index = 0; w_index < window_size; ++w_index ) {
         sh_samples[threadIdx.x * window_size + w_index] = samples[index + w_index];
         sh_window_sums[w_index] = make_hipFloatComplex( 0.f, 0.f );
      }
      __syncthreads();
      
      for( int w_index = 0; w_index < window_size; ++w_index ) {
         sh_window_sums[threadIdx.x] += sh_samples[threadIdx.x * window_size + w_index]
      }
      __syncthreads();

      window_sums[index] = sh_window_sums[threadIdx.x]
   }

}
