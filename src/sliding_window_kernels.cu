#include "hip/hip_runtime.h"
#include "sliding_window_kernels.cuh"

//////////////////////////////////////
// Calculate sliding window sums
//////////////////////////////////////
__global__ void sliding_window_original(
   hipfftComplex* __restrict__ window_sums, 
   hipfftComplex* const __restrict__ samples, 
   const int window_size, 
   const int num_windowed_samples ) {
   
   // Assuming one stream
   int global_index = 1*(blockIdx.x * blockDim.x) + 1*(threadIdx.x);
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;
   for ( int index = global_index; index < num_windowed_samples; index+=stride ) {
      /*hipfftComplex t_window_sum = make_hipFloatComplex(0.f, 0.f);*/
      hipfftComplex t_window_sum = samples[index];

      /*for ( int w_index = 0; w_index < window_size; ++w_index ) {*/
      for ( int w_index = 1; w_index < window_size; ++w_index ) {
         /*t_window_sum = make_hipFloatComplex( ( t_window_sum.x + samples[index + w_index].x ),*/
         /*   ( t_window_sum.y + samples[index + w_index].y ) );*/
         t_window_sum = hipCaddf( t_window_sum, samples[index + w_index] );
      }

      /*window_sums[index] = make_hipFloatComplex( t_window_sum.x, t_window_sum.y );*/
      window_sums[index] = t_window_sum;
   }
} // end of __global__ void sliding_window_original

__global__ void sliding_window_vectorized_loads(
   hipfftComplex* __restrict__ window_sums, 
   hipfftComplex* const __restrict__ samples, 
   const int window_size, 
   const int num_windowed_samples ) {
   
   // Assuming one stream
   int global_index = blockIdx.x * blockDim.x + threadIdx.x;
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;
   for ( int index = global_index; index < num_windowed_samples; index+=stride ) {
      /*hipfftComplex t_window_sum = make_hipFloatComplex(0.f, 0.f);*/
      hipfftComplex t_window_sum = samples[index];

      /*for ( int w_index = 0; w_index < window_size; ++w_index ) {*/
      for ( int w_index = 1; w_index < window_size; w_index+=2 ) {
         /*t_window_sum = make_hipFloatComplex( ( t_window_sum.x + samples[index + w_index].x ),*/
         /*   ( t_window_sum.y + samples[index + w_index].y ) );*/
         t_window_sum = hipCaddf( t_window_sum, samples[index + w_index] );
         t_window_sum = hipCaddf( t_window_sum, samples[index + w_index + 1] );
      }

      /*window_sums[index] = make_hipFloatComplex( t_window_sum.x, t_window_sum.y );*/
      window_sums[index] = t_window_sum;
   }
} // end of __global__ void sliding_window_original

// Shared Memory Implementation
__global__ void sliding_window_sh_mem( hipfftComplex* __restrict__ window_sums, 
   hipfftComplex* const __restrict__ samples, 
   const int window_size, 
   const int num_windowed_samples ) {

   // Assuming one stream
   int global_index = 1*(blockIdx.x * blockDim.x) + 1*(threadIdx.x);
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;

   __shared__ hipfftComplex sh_samples[6144];

   for ( int index = global_index; index < num_windowed_samples; index+=stride ) {
      
      for( int w_index = 0; w_index < window_size; ++w_index ) {
         sh_samples[threadIdx.x * window_size + w_index] = samples[index + w_index];
      }
      __syncthreads();
      
      for( int w_index = 0; w_index < window_size; ++w_index ) {
         sh_samples[blockIdx.x * window_size] = hipCaddf( sh_samples[blockIdx.x * window_size], 
            sh_samples[threadIdx.x * window_size + w_index] );
      }
      __syncthreads();

      if (threadIdx.x == 0) {
         window_sums[index] = sh_samples[blockIdx.x * window_size];
      }
   }

}
