#include "hip/hip_runtime.h"
#include "sliding_window_kernels.cuh"

//////////////////////////////////////
// Calculate sliding window sums
//////////////////////////////////////
__global__ void sliding_window_original(
   hipfftComplex* __restrict__ window_sums, 
   hipfftComplex* const __restrict__ samples, 
   const int window_size, 
   const int num_windowed_samples ) {
   
   // Assuming one stream
   int global_index = 1*(blockIdx.x * blockDim.x) + 1*(threadIdx.x);
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;
   for ( int index = global_index; index < num_windowed_samples; index+=stride ) {
      hipfftComplex t_window_sum = make_hipFloatComplex(0.f, 0.f);

      for ( int w_index = 0; w_index < window_size; ++w_index ) {
         t_window_sum = make_hipFloatComplex( ( t_window_sum.x + samples[index + w_index].x ),
            ( t_window_sum.y + samples[index + w_index].y ) );
      }

      window_sums[index] = make_hipFloatComplex( t_window_sum.x, t_window_sum.y );
   }
} // end of __global__ void sliding_window_original


// Shared Memory Implementation
__global__ void sliding_window_sh_mem( hipfftComplex* __restrict__ window_sums, 
   hipfftComplex* const __restrict__ samples, 
   const int window_size, 
   const int num_windowed_samples ) {

   // Assuming one stream
   int global_index = 1*(blockIdx.x * blockDim.x) + 1*(threadIdx.x);
   // stride is set to the total number of threads in the grid
   int stride = blockDim.x * gridDim.x;

   extern __shared__ hipfftComplex sh_samples[];

   for( int index = global_index; index < num_windowed_samples; index+=stride ) {
      
      for( int w_index = 0; w_index < window_size; ++w_index ) {
         sh_samples[window_size * threadIdx.x + w_index] = samples[ global_index + w_index ];
      }
      __syncthreads();

      for( int w_index = 1; w_index < window_size; ++w_index ) {
         sh_samples[threadIdx.x * window_size] = hipCaddf( sh_samples[threadIdx.x * window_size], sh_samples[threadIdx.x * window_size + w_index] );
         __syncthreads();
      }

      window_sums[global_index] = sh_samples[threadIdx.x * window_size];
   }
}
